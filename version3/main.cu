#include "hip/hip_runtime.h"
#include "CUDASupport.h"

#include "radixsort.cuh"

#include <algorithm>
#include <cstring> //memset
#include <time.h>

void printArray(unsigned int* const h_input, const size_t numElems)
 {
	for (unsigned int i = 0; i < numElems; ++i)
		printf("%d  ", h_input[i]);
	printf("\n");
 }

__device__
 void dev_printArray(unsigned int* const d_input, const size_t numElems)
 {
	for (unsigned int i = 0; i < numElems; ++i)
		printf("%d \t", d_input[i]);
	printf("\n");
 }

int main()
{
	// srand(time(NULL));
	const dim3 blockSize(64);
	const size_t numElems = 10000000;
	const unsigned int numBits = 3;

	unsigned int* h_inputVals = (unsigned int*) malloc(sizeof(unsigned int) * numElems);
	for (int i = 0; i < numElems; ++i)
	{
		h_inputVals[i] = rand() % 1000000000 + 1;
	}
	
	// printArray(h_inputVals, numElems);

	unsigned int* d_inputVals;
	checkCudaErrors(hipMalloc(&d_inputVals, sizeof(unsigned int) * numElems));
	checkCudaErrors(hipMemcpy(d_inputVals, h_inputVals, sizeof(unsigned int) * numElems, hipMemcpyHostToDevice));

	unsigned int* d_outputVals;
	checkCudaErrors(hipMalloc(&d_outputVals, sizeof(unsigned int) * numElems));

	clock_t begin = clock();
	radix_sort(h_inputVals, d_inputVals, d_outputVals, numElems, numBits, blockSize);
	clock_t end = clock();
	double duration = end - begin;
	printf("Time: %lf\n", duration);

	unsigned int* h_outputVals = (unsigned int*) malloc(sizeof(unsigned int) * numElems);
	checkCudaErrors(hipMemcpy(h_outputVals, d_outputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));

	// printArray(h_outputVals, numElems);
	bool check = true;
	for (int i = 1; i < numElems; ++i)
	{
		if (h_outputVals[i] < h_outputVals[i - 1])
		{
			printf("\nfalse at index : %d\n", i);
			check = false;
			break;
		}
	}

	if (check)
		printf("\nTRUE\n");
	else
		printf("\nFALSE\n");


	
	free(h_inputVals);
	free(h_outputVals);
	checkCudaErrors(hipFree(d_inputVals));
	checkCudaErrors(hipFree(d_outputVals));
	return 0;
}
