#include "hip/hip_runtime.h"
#include "scatter.cuh"

__global__
void scatter(unsigned int* const d_inputVals,
			unsigned int* const d_outputVals,
			const unsigned int* const d_histScan,
			const size_t numElems,
			const size_t numBins,
			const unsigned int mask,
			const unsigned int digitOrder)
{
	extern __shared__ unsigned int s_blkIn[]; // blockSize

	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= numElems)
		return;

	//load data
	s_blkIn[threadIdx.x] = d_inputVals[idx];
	__syncthreads();

	//get number of elements ( < this element and = this element but locate on prev block 
	unsigned int bin = (s_blkIn[threadIdx.x] & mask) >> digitOrder;
	unsigned int numPrevElems = d_histScan[bin * gridDim.x + blockIdx.x];

	//calc number of element = this element on this block
	unsigned int count = 0;
	for (int i = 0; i < threadIdx.x; ++i)
	{
		unsigned int prevBin = (s_blkIn[i] & mask) >> digitOrder;
		if (bin == prevBin)
			++count;
	}
	
	// scatter to result array
	unsigned int rank = numPrevElems + count;
	d_outputVals[rank] = s_blkIn[threadIdx.x];
	// printf("block - idx - rank - val: %d - %d - %d - %d\n", blockIdx.x, idx, rank, s_blkIn[threadIdx.x]);

}

void host_scatter(unsigned int* const d_inputVals,
				unsigned int* const d_outputVals,
				const size_t numElems,
				const size_t numBins,
				const unsigned int* const d_histScan,
				const unsigned int mask,
				const unsigned int digitOrder,
				const dim3 blockSize)
{
	const dim3 gridSize((numElems - 1) / blockSize.x + 1);

	unsigned int sharedSize = blockSize.x * sizeof(unsigned int);
	scatter<<<gridSize, blockSize, sharedSize>>> (d_inputVals, d_outputVals, d_histScan, numElems,numBins, mask, digitOrder);
}